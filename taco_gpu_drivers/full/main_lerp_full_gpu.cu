#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>
#include <time.h>
#include "taco.h"

extern taco_tensor_t* init_taco_tensor(int32_t order, int32_t csize, int32_t* dimensions);
extern taco_tensor_t* init_taco_tensor_gpu(taco_tensor_t* ht);
extern void fill_array(float* arr, int len);
extern double calc_spent_time(struct timespec end, struct timespec start);
extern double average(double* values, int len);
extern void gpuAssert(hipError_t code, const char *file, int line, bool abort);
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

#ifndef TACO_TENSOR_T_DEFINED
#define TACO_TENSOR_T_DEFINED
typedef enum { taco_mode_dense, taco_mode_sparse } taco_mode_t;
typedef struct {
    int32_t      order;
    int32_t*     dimensions;
    int32_t      csize;
    int32_t*     mode_ordering;
    taco_mode_t* mode_types;
    uint8_t***   indices;
    float*       vals;
    uint8_t*     fill_value;
    int32_t      vals_size;
} taco_tensor_t;
#endif

__global__
void computeDeviceKernel0(taco_tensor_t * __restrict__ a, float &b_val, taco_tensor_t * __restrict__ c, taco_tensor_t * __restrict__ d){
  float* __restrict__ a_vals = (float*)(a->vals);
  float* __restrict__ c_vals = (float*)(c->vals);
  int d1_dimension = (int)(d->dimensions[0]);
  float* __restrict__ d_vals = (float*)(d->vals);

  int32_t i65 = blockIdx.x;
  int32_t i66 = (threadIdx.x % (256));
  if (threadIdx.x >= 256) {
    return;
  }

  int32_t i = i65 * 256 + i66;
  if (i >= d1_dimension)
    return;

  a_vals[i] = b_val * c_vals[i] + (1 - b_val) * d_vals[i];
}

int compute(taco_tensor_t *a, taco_tensor_t *b, taco_tensor_t *c, taco_tensor_t *d) {
  float* __restrict__ b_vals = (float*)(b->vals);
  int d1_dimension = (int)(d->dimensions[0]);

  float* b_val_ptr;
  gpuErrchk(hipMallocManaged((void**)&b_val_ptr, sizeof(float)));
  float& b_val = *b_val_ptr;
  b_val = b_vals[0];

  computeDeviceKernel0<<<((d1_dimension + 255) / 256), 256>>>(a, b_val, c, d);
  hipDeviceSynchronize();
  return 0;
}


int main(int argc, char* argv[]) {
    int n_runs = atoi(argv[1]);
    if (argc < 2) {
        printf("Please specify number of executions!\n");
        exit(1);
    }

    srand(time(0));
    struct timespec start, end_orig, end_taco;
    double* orig_run_times = (double*)malloc(n_runs * sizeof(double));
    double* taco_run_times = (double*)malloc(n_runs * sizeof(double));

    int N = 1000000;
    float* a = (float*)malloc(N * sizeof(float));
    float* b = (float*)malloc(N * sizeof(float));
    float* c = (float*)malloc(N * sizeof(float));
    float t = 0.5f;

    int dims[1] = {N};
    taco_tensor_t* a_tt = init_taco_tensor(1, sizeof(float), dims);
    a_tt->vals = a;
    a_tt = init_taco_tensor_gpu(a_tt);

    taco_tensor_t* b_tt = init_taco_tensor(1, sizeof(float), dims);
    b_tt->vals = b;
    b_tt = init_taco_tensor_gpu(b_tt);

    taco_tensor_t* c_tt = init_taco_tensor(1, sizeof(float), dims);
    c_tt->vals = c;
    c_tt = init_taco_tensor_gpu(c_tt);

    int t_dims[1] = {1};
    taco_tensor_t* t_tt = init_taco_tensor(1, sizeof(float), t_dims);
    t_tt->vals = &t;
    t_tt = init_taco_tensor_gpu(t_tt);

    for (int i = 0; i < n_runs; i++) {
        fill_array(a, N);
        fill_array(b, N);
        fill_array(c, N);

        clock_gettime(CLOCK_MONOTONIC, &start);
        lerp(out, x, y, alpha, n);
        clock_gettime(CLOCK_MONOTONIC, &end_orig);

        compute(a_tt, b_tt, c_tt, t_tt);
        clock_gettime(CLOCK_MONOTONIC, &end_taco);

        orig_run_times[i] = calc_spent_time(end_orig, start);
        taco_run_times[i] = calc_spent_time(end_taco, end_orig);
    }

    double orig_time = average(orig_run_times, n_runs);
    double taco_time = average(taco_run_times, n_runs);
    printf("%.5lf %.5lf\n", orig_time, taco_time);

    free(orig_run_times);
    free(taco_run_times);
    free(a);
    free(b);
    free(c);
    
    return 0;
}

