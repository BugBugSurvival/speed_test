#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "taco.h"
#include <hip/hip_runtime.h>

taco_tensor_t* init_taco_tensor(int32_t order, int32_t csize, int32_t* dimensions){
  taco_tensor_t* t = (taco_tensor_t*)malloc(sizeof(taco_tensor_t));
  t->order = order;
  t->dimensions = dimensions;

  return t;
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

taco_tensor_t* init_taco_tensor_gpu(taco_tensor_t* ht) {
  taco_tensor_t* t = NULL;
  gpuErrchk(hipMallocManaged((void**)&t, sizeof(taco_tensor_t)));
  t->order = ht->order;
  gpuErrchk(hipMallocManaged((void**)&(t->dimensions), t->order * sizeof(int32_t)));
  t->vals_size = 1;
  for(int i = 0; i < t->order; i++){
    t->dimensions[i] = ht->dimensions[i];
    t->vals_size *= t->dimensions[i];
  }
  gpuErrchk(hipMallocManaged((void**)&(t->vals), t->vals_size * sizeof(float)));
  int* d_t_vals;
  float* ht_vals = (float*)ht->vals;
  gpuErrchk(hipMallocManaged((void**)&d_t_vals, t->vals_size * sizeof(float)));
  gpuErrchk(hipMemcpy(d_t_vals, ht_vals, t->vals_size * sizeof(float), hipMemcpyHostToDevice));
  t->vals=(float*)d_t_vals;
  return t;
}

double calc_spent_time(struct timespec end, struct timespec start){
  double result = end.tv_sec - start.tv_sec;
  result += (end.tv_nsec - start.tv_nsec) / 1000000000.0;
  return result;
}

double average(double* values, int len){
  double average = 0.0;
  for(int i = 0; i < len; i++) average += values[i];
    return average / len;
}

void fill_array(float* arr, int len){
  for(int i = 0; i < len; i++)
    arr[i] = 1 + ((float)rand()) / RAND_MAX;
}

void print_array(const char* name, float* arr, int len){
  printf("%s: %f: ", name, len);
  for(int i = 0; i < len; i++)
    printf("%f ", arr[i]);
  printf("\n");
}




